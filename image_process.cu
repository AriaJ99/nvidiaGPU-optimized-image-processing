#include "hip/hip_runtime.h"
#include "image_process.cuh"

cv::Mat run_module(cv::Mat input_image,const int &extra_info,const std::string &module){
    cv::Mat output_image;
    cv::Mat temp_image;
    //image dims
    //std::cout<<(int)input_image.data[5]<<std::endl;
    int width=input_image.cols;
    int height=input_image.rows;
    int RGB_size=width*height*3*sizeof(unsigned char);
    int grayscale_size=width*height*sizeof(unsigned char);
    bool is_RGB=input_image.channels();
    //GPU thread dims
    dim3 block_size(32,32);
    dim3 grid_size((width+block_size.x-1)/block_size.x,(height+block_size.y-1)/block_size.y);
    //device memory prepration
    unsigned char *device_input,*device_output;
    //matching the module
    if(module=="RGB_to_grayscale"){
        std::cout<<"converting to grayscale"<<std::endl;
        mem_init(device_input,RGB_size,device_output,grayscale_size,input_image);

        RGB_to_grayscale_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");

        output_image=mem_to_image(device_output,height,width,grayscale_size,false);

        std::cout<<"convertion to grayscale done!"<<std::endl;
    }
    else if(module=="blur"){
        std::cout<<"start blurring"<<std::endl;

        int kernel_size=extra_info;

        if(is_RGB)
            mem_init(device_input,RGB_size,device_output,RGB_size,input_image);
        else
            mem_init(device_input,grayscale_size,device_output,grayscale_size,input_image);

        blur_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height,kernel_size,is_RGB);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
        
        output_image=mem_to_image(device_output,height,width,grayscale_size,is_RGB);

        std::cout<<"image has been blurred!"<<std::endl;
    }
    else if(module=="edge_detection"){
        int kernel_size=extra_info;
        //temporary init as it only covers 3*3 kernels now
        kernel_size=3;
        std::cout<<"start edge detection"<<std::endl;
        temp_image=input_image;
        //convert to grayscale if it's RGB
        if(!is_RGB){
            std::cout<<"converting to grayscale"<<std::endl;

            mem_init(device_input,RGB_size,device_output,grayscale_size,input_image);

            RGB_to_grayscale_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height);
            kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
            
            temp_image=mem_to_image(device_output,height,width,grayscale_size,false);

            kernel_error_checker(hipFree(device_input),"Free");
            kernel_error_checker(hipFree(device_output),"Free");
            std::cout<<"convertion to grayscale done!"<<std::endl;
        }
        std::cout<<"applying the threshold"<<std::endl;
        mem_init(device_input,grayscale_size,device_output,grayscale_size,temp_image);

        edge_detection_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height,kernel_size);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");

        output_image=mem_to_image(device_output,height,width,grayscale_size,false);
        std::cout<<"edge detection has been done!"<<std::endl;
    }
    else if(module=="threshold"){
        int threshold=128; 
        if(0<=extra_info && extra_info<=255){
            threshold=extra_info;
            std::cout<<"Threshold value: "<<threshold<<std::endl;
        }
        temp_image=input_image;
        //convert to grayscale if it's RGB
        if(!is_RGB){
            std::cout<<"converting to grayscale"<<std::endl;

            mem_init(device_input,RGB_size,device_output,grayscale_size,input_image);

            RGB_to_grayscale_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height);
            kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
            
            temp_image=mem_to_image(device_output,height,width,grayscale_size,false);

            kernel_error_checker(hipFree(device_input),"Free");
            kernel_error_checker(hipFree(device_output),"Free");
            std::cout<<"convertion to grayscale done!"<<std::endl;
        }
        std::cout<<"applying the threshold"<<std::endl;
        mem_init(device_input,grayscale_size,device_output,grayscale_size,temp_image);

        thresholding_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height,threshold);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");

        output_image=mem_to_image(device_output,height,width,grayscale_size,false);
        std::cout<<"Threshold has been applied!"<<std::endl;
    }
    else if(module=="adjust_bright"){
        std::cout<<"start adjusting brightness"<<std::endl;

        int offset=extra_info;

        if(is_RGB)
            mem_init(device_input,RGB_size,device_output,RGB_size,input_image);
        else
            mem_init(device_input,grayscale_size,device_output,grayscale_size,input_image);

        brightness_adjustment_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height,offset,is_RGB);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
        
        output_image=mem_to_image(device_output,height,width,grayscale_size,is_RGB);

        std::cout<<"brightness has been adjusted!"<<std::endl;
    }
    else{
        std::cerr<<"Module not found!"<<std::endl;
        exit(-2);
    }
    hipFree(device_input);
    hipFree(device_output);
    return output_image;
}
void kernel_error_checker(hipError_t err, std::string method){
    if (err!=hipSuccess){
        std::cerr<<"CUDA "<<method<<" error: "<<hipGetErrorString(err)<<std::endl;
        exit(-1);
    }
}
void mem_init(unsigned char *&input,int input_size,unsigned char *&output,int output_size,cv::Mat &image){
    kernel_error_checker(hipMalloc((void**)&input,input_size),"Malloc");
    kernel_error_checker(hipMalloc((void**)&output,output_size),"Malloc");
    kernel_error_checker(hipMemcpy(input,image.data,input_size,hipMemcpyHostToDevice),"Memcpy");
}
cv::Mat mem_to_image(unsigned char *input,int height,int width,int size, bool is_RGB){
    cv::Mat temp_img;

    if(is_RGB){
        temp_img=cv::Mat(height,width,CV_8UC3);
        kernel_error_checker(hipMemcpy(temp_img.data,input,size*3,hipMemcpyDeviceToHost),"Memcpy");
    }
    else{
        temp_img=cv::Mat(height,width,CV_8UC1);
        kernel_error_checker(hipMemcpy(temp_img.data,input,size,hipMemcpyDeviceToHost),"Memcpy");
    }
    
    return temp_img;
}
__device__ int2 find_index() {
    return make_int2(
        blockIdx.x*blockDim.x+threadIdx.x,
        blockIdx.y*blockDim.y+threadIdx.y
    );
}
__global__ void RGB_to_grayscale_kernel(unsigned char *input,unsigned char *output,int width,int height){
    int2 idx=find_index();
    if(idx.x<width && idx.y<height){
        int rgb_img_idx=(idx.y*width+idx.x)*3;
        int gray_img_idx=(idx.y*width+idx.x);
        unsigned char r=input[rgb_img_idx];
        unsigned char g=input[rgb_img_idx+1];
        unsigned char b=input[rgb_img_idx+2];
        //calculate combination of RGB based on human perception of different channels
        output[gray_img_idx]=static_cast<unsigned char>(0.21f*r+0.72f*g+0.07f*b);
    }

}
__global__ void blur_kernel(unsigned char *input,unsigned char *output,int width,int height,int kernel_size,bool is_RGB){
    //TODO handle error for even sized kernels 
    int2 idx=find_index();
    __shared__ float kernel[100];
    //kernel construction
    if(idx.x==0 && idx.y==0){
        if(kernel_size%2==1){ 
            for(int i=0;i<kernel_size*kernel_size;i++)
                kernel[i]=1.0/(kernel_size*kernel_size);
        }   
    }
    __syncthreads();
    if(idx.x<width && idx.y<height){
        image_kernel_convolution(input,output,width,height,idx.x,idx.y,kernel_size,kernel,is_RGB);
    }
    
}
__global__ void thresholding_kernel(unsigned char *input,unsigned char *output,int width,int height,int threshold){
    int2 idx=find_index();
    if(idx.x<width && idx.y<height){
        int grayscale_img_idx=(idx.y*width+idx.x);
        if(static_cast<unsigned char>(threshold)<=input[grayscale_img_idx])
            output[grayscale_img_idx]=static_cast<unsigned char>(255);
        else
            output[grayscale_img_idx]=static_cast<unsigned char>(0);
    }
}
__global__ void brightness_adjustment_kernel(unsigned char *input,unsigned char *output,int width,int height,int offset,bool is_RGB){
    int2 idx=find_index();
    if(is_RGB){
        int RGB_img_idx=(idx.y*width+idx.x)*3;
        if(idx.x<width && idx.y<height){
                int grayscale_img_idx=(idx.y*width+idx.x);
                if(offset>=0){
                    output[RGB_img_idx]=min(input[RGB_img_idx]+offset,255);
                    output[RGB_img_idx+1]=min(input[RGB_img_idx+1]+offset,255);
                    output[RGB_img_idx+2]=min(input[RGB_img_idx+2]+offset,255);
                }
                else{
                    output[RGB_img_idx]=max(input[RGB_img_idx]-offset,0);
                    output[RGB_img_idx+1]=max(input[RGB_img_idx+1]-offset,0);
                    output[RGB_img_idx+2]=max(input[RGB_img_idx+2]-offset,0);
                }
            }
    }
    else{
        int grayscale_img_idx=idx.y*width+idx.x;

        if(idx.x<width && idx.y<height){
                int grayscale_img_idx=(idx.y*width+idx.x);
                if(offset>=0)
                    output[grayscale_img_idx]=min(input[grayscale_img_idx]+offset,255);
                else
                    output[grayscale_img_idx]=max(input[grayscale_img_idx]-offset,0);

            }
    }
}
__device__ void image_kernel_convolution(unsigned char *input,unsigned char *output,int width,int height,int x,int y,int kernel_size,int kernel_weight,float *kernel,bool is_RGB){
    //TODO pixel overflow handling

    int RGB_idx=(y*width+x)*3;
    int grayscale_idx=y*width+x;

    float sum_r=0.0,sum_g=0.0,sum_b=0.0,sum_gray=0.0;
    for(int dy=0;dy<kernel_size;dy++)
        for(int dx=0;dx<kernel_size;dx++){
            int nx=x+dx-kernel_size/2;
            int ny=y+dy-kernel_size/2;

            if(nx>=0 && nx<width && ny>=0 && ny<height){
                float weight=kernel[dy*kernel_size+dx];
                if(is_RGB){
                    int nidx=(ny*width+nx)*3;

                    sum_r+=input[nidx];
                    sum_g+=input[nidx+1];
                    sum_b+=input[nidx+2];
                }
                else{
                    int nidx=ny*width+nx;

                    sum_gray+=input[nidx];
                }
                
            }
        }
    if(is_RGB){
        output[RGB_idx]=static_cast<unsigned char>(sum_r/kernel_weight);
        output[RGB_idx+1]=static_cast<unsigned char>(sum_g/kernel_weight);
        output[RGB_idx+2]=static_cast<unsigned char>(sum_b/kernel_weight);
    }
    else{
        output[grayscale_idx]=static_cast<unsigned char>(sum_gray/kernel_weight);
    }    

}
__global__ void edge_detection_kernel(unsigned char *input,unsigned char *output,int width,int height,int kernel_size){
    //TODO handle error for even sized kernels 
    int2 idx=find_index();
    __shared__ float kernel[9];
    //kernel construction
    if(idx.x==0 && idx.y==0){
        if(kernel_size%2==1){
            //kernel values
            //x=0,y=0
            kernel[0]=0;
            //x=1,y=0
            kernel[1]=-1;
            //x=2,y=0
            kernel[2]=0;
            //x=0,y=1
            kernel[3]=-1;
            //x=1,y=1
            kernel[4]=4;
            //x=2,y=1
            kernel[5]=-1;
            //x=0,y=2
            kernel[6]=0;
            //x=1,y=2
            kernel[7]=-1;
            //x=2,y=2
            kernel[8]=0;
        }
    }
    __syncthreads();

    if(idx.x<width && idx.y<height){
        image_kernel_convolution(input,output,width,height,idx.x,idx.y,kernel_size,kernel,false);
    }

}