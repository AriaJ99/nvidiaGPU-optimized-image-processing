#include "hip/hip_runtime.h"
#include "image_process.cuh"

cv::Mat run_module(cv::Mat input_image,const std::string &extra_info,const std::string &module){
    cv::Mat output_image;
    cv::Mat temp_image;
    //image dims
    //std::cout<<(int)input_image.data[5]<<std::endl;
    int width=input_image.cols;
    int height=input_image.rows;
    int RGB_size=width*height*3*sizeof(unsigned char);
    int grayscale_size=width*height*sizeof(unsigned char);
    bool is_RGB=input_image.channels();
    //GPU thread dims
    dim3 block_size(32,32);
    dim3 grid_size((width+block_size.x-1)/block_size.x,(height+block_size.y-1)/block_size.y);
    //device memory prepration
    unsigned char *device_input,*device_output;
    //matching the module
    if(module=="RGB_to_grayscale"){
        std::cout<<"converting to grayscale"<<std::endl;
        mem_init(device_input,RGB_size,device_output,grayscale_size,input_image);

        RGB_to_grayscale_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");

        output_image=mem_to_image(device_output,height,width,grayscale_size,false);

        std::cout<<"convertion to grayscale done!"<<std::endl;
    }
    else if(module=="blur"){
        std::cout<<"start blurring"<<std::endl;

        int kernel_size=std::stoi(extra_info);

        if(is_RGB)
            mem_init(device_input,RGB_size,device_output,RGB_size,input_image);
        else
            mem_init(device_input,grayscale_size,device_output,grayscale_size,input_image);

        blur_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height,kernel_size,is_RGB);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
        
        output_image=mem_to_image(device_output,height,width,grayscale_size,is_RGB);

        std::cout<<"image has been blurred!"<<std::endl;
    }
    else if(module=="edge_detection"){
        
        int method;

        if(extra_info=="sobel")
            method=SOBEL;
        else if(extra_info=="prewitt")
            method=PREWITT;
        else if(extra_info=="robert")
            method=ROBERT;
        else
            method=SOBEL;


        std::cout<<"start edge detection"<<std::endl;
        temp_image=input_image;

        unsigned char *vertical_device,*horizontal_device;
        //convert to grayscale if it's RGB
        if(is_RGB){
            std::cout<<"converting to grayscale"<<std::endl;

            mem_init(device_input,RGB_size,device_output,grayscale_size,input_image);

            RGB_to_grayscale_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height);
            kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
            
            temp_image=mem_to_image(device_output,height,width,grayscale_size,false);

            kernel_error_checker(hipFree(device_input),"Free");
            kernel_error_checker(hipFree(device_output),"Free");
            std::cout<<"convertion to grayscale done!"<<std::endl;
        }

        mem_init(device_input,grayscale_size,device_output,grayscale_size,temp_image);

        kernel_error_checker(hipMalloc((void**)&vertical_device,grayscale_size),"Malloc");
        kernel_error_checker(hipMalloc((void**)&horizontal_device,grayscale_size),"Malloc");
        

        edge_detection_kernel<<<grid_size,block_size>>>(device_input,vertical_device,width,height,1,method);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
        
        edge_detection_kernel<<<grid_size,block_size>>>(device_input,horizontal_device,width,height,2,method);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
        
        matrix_magnitude_kernel<<<grid_size,block_size>>>(device_output,vertical_device,horizontal_device,width,height);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");

        output_image=mem_to_image(device_output,height,width,grayscale_size,false);
        std::cout<<"edge detection has been done!"<<std::endl;
    }
    else if(module=="threshold"){
        int threshold=std::stoi(extra_info);
        if(!(0<=threshold && threshold<=255))
            threshold=128;

        std::cout<<"Threshold value: "<<threshold<<std::endl;

        temp_image=input_image;
        //convert to grayscale if it's RGB
        if(is_RGB){
            std::cout<<"converting to grayscale"<<std::endl;

            mem_init(device_input,RGB_size,device_output,grayscale_size,input_image);

            RGB_to_grayscale_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height);
            kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
            
            temp_image=mem_to_image(device_output,height,width,grayscale_size,false);

            kernel_error_checker(hipFree(device_input),"Free");
            kernel_error_checker(hipFree(device_output),"Free");
            std::cout<<"convertion to grayscale done!"<<std::endl;
        }
        std::cout<<"applying the threshold"<<std::endl;
        mem_init(device_input,grayscale_size,device_output,grayscale_size,temp_image);

        thresholding_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height,threshold);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");

        output_image=mem_to_image(device_output,height,width,grayscale_size,false);
        std::cout<<"Threshold has been applied!"<<std::endl;
    }
    else if(module=="adjust_bright"){
        std::cout<<"start adjusting brightness"<<std::endl;

        int offset=std::stoi(extra_info);

        if(is_RGB)
            mem_init(device_input,RGB_size,device_output,RGB_size,input_image);
        else
            mem_init(device_input,grayscale_size,device_output,grayscale_size,input_image);

        brightness_adjustment_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height,offset,is_RGB);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
        
        output_image=mem_to_image(device_output,height,width,grayscale_size,is_RGB);

        std::cout<<"brightness has been adjusted!"<<std::endl;
    }
    else if(module=="edge_sharpening"){
        
        std::cout<<"start edge sharpening"<<std::endl;
 
        if(is_RGB)
            mem_init(device_input,RGB_size,device_output,RGB_size,input_image);
        else
            mem_init(device_input,grayscale_size,device_output,grayscale_size,input_image);

        edge_sharpening_kernel<<<grid_size,block_size>>>(device_input,device_output,width,height,is_RGB);
        kernel_error_checker(hipDeviceSynchronize(),"DeviceSynchronize");
        
        output_image=mem_to_image(device_output,height,width,grayscale_size,is_RGB);

        std::cout<<"edge sharpening has been done!"<<std::endl;
    }
    else{
        std::cerr<<"Module not found!"<<std::endl;
        exit(-2);
    }
    hipFree(device_input);
    hipFree(device_output);
    return output_image;
}
void kernel_error_checker(hipError_t err, std::string method){
    if (err!=hipSuccess){
        std::cerr<<"CUDA "<<method<<" error: "<<hipGetErrorString(err)<<std::endl;
        exit(-1);
    }
}
void mem_init(unsigned char *&input,int input_size,unsigned char *&output,int output_size,cv::Mat &image){
    kernel_error_checker(hipMalloc((void**)&input,input_size),"Malloc");
    kernel_error_checker(hipMalloc((void**)&output,output_size),"Malloc");
    kernel_error_checker(hipMemcpy(input,image.data,input_size,hipMemcpyHostToDevice),"Memcpy");
}
cv::Mat mem_to_image(unsigned char *input,int height,int width,int size, bool is_RGB){
    cv::Mat temp_img;

    if(is_RGB){
        temp_img=cv::Mat(height,width,CV_8UC3);
        kernel_error_checker(hipMemcpy(temp_img.data,input,size*3,hipMemcpyDeviceToHost),"Memcpy");
    }
    else{
        temp_img=cv::Mat(height,width,CV_8UC1);
        kernel_error_checker(hipMemcpy(temp_img.data,input,size,hipMemcpyDeviceToHost),"Memcpy");
    }
    
    return temp_img;
}
__device__ int2 find_index() {
    return make_int2(
        blockIdx.x*blockDim.x+threadIdx.x,
        blockIdx.y*blockDim.y+threadIdx.y
    );
}
__global__ void RGB_to_grayscale_kernel(unsigned char *input,unsigned char *output,int width,int height){
    int2 idx=find_index();
    if(idx.x<width && idx.y<height){
        int rgb_img_idx=(idx.y*width+idx.x)*3;
        int gray_img_idx=(idx.y*width+idx.x);
        unsigned char r=input[rgb_img_idx];
        unsigned char g=input[rgb_img_idx+1];
        unsigned char b=input[rgb_img_idx+2];
        //calculate combination of RGB based on human perception of different channels
        output[gray_img_idx]=static_cast<unsigned char>(0.21f*r+0.72f*g+0.07f*b);
    }

}
__global__ void blur_kernel(unsigned char *input,unsigned char *output,int width,int height,int kernel_size,bool is_RGB){
    //TODO handle error for even sized kernels 
    int2 idx=find_index();
    __shared__ float kernel[625];
    //kernel construction
    for(int i=0;i<kernel_size*kernel_size;i++)
        kernel[i]=1.0;
    
    __syncthreads();

    if(idx.x<width && idx.y<height){
        image_kernel_convolution(input,output,width,height,idx.x,idx.y,kernel_size,kernel_size*kernel_size,kernel,is_RGB);
    }
    
}
__global__ void thresholding_kernel(unsigned char *input,unsigned char *output,int width,int height,int threshold){
    int2 idx=find_index();
    if(idx.x<width && idx.y<height){
        int grayscale_img_idx=(idx.y*width+idx.x);
        if(static_cast<unsigned char>(threshold)<=input[grayscale_img_idx])
            output[grayscale_img_idx]=static_cast<unsigned char>(255);
        else
            output[grayscale_img_idx]=static_cast<unsigned char>(0);
    }
}
__global__ void brightness_adjustment_kernel(unsigned char *input,unsigned char *output,int width,int height,int offset,bool is_RGB){
    int2 idx=find_index();
    if(is_RGB){
        int RGB_img_idx=(idx.y*width+idx.x)*3;
        if(idx.x<width && idx.y<height){
                int grayscale_img_idx=(idx.y*width+idx.x);
                if(offset>=0){
                    output[RGB_img_idx]=min(input[RGB_img_idx]+offset,255);
                    output[RGB_img_idx+1]=min(input[RGB_img_idx+1]+offset,255);
                    output[RGB_img_idx+2]=min(input[RGB_img_idx+2]+offset,255);
                }
                else{
                    output[RGB_img_idx]=max(input[RGB_img_idx]-offset,0);
                    output[RGB_img_idx+1]=max(input[RGB_img_idx+1]-offset,0);
                    output[RGB_img_idx+2]=max(input[RGB_img_idx+2]-offset,0);
                }
            }
    }
    else{
        int grayscale_img_idx=idx.y*width+idx.x;

        if(idx.x<width && idx.y<height){
                int grayscale_img_idx=(idx.y*width+idx.x);
                if(offset>=0)
                    output[grayscale_img_idx]=min(input[grayscale_img_idx]+offset,255);
                else
                    output[grayscale_img_idx]=max(input[grayscale_img_idx]-offset,0);

            }
    }
}
__device__ void image_kernel_convolution(unsigned char *input,unsigned char *output,int width,int height,int x,int y,int kernel_size,int kernel_weight,float *kernel,bool is_RGB){
    //TODO pixel overflow handling

    int RGB_idx=(y*width+x)*3;
    int grayscale_idx=y*width+x;

    float sum_r=0.0,sum_g=0.0,sum_b=0.0,sum_gray=0.0;
    for(int dy=0;dy<kernel_size;dy++)
        for(int dx=0;dx<kernel_size;dx++){
            int nx=x+dx-kernel_size/2;
            int ny=y+dy-kernel_size/2;

            if(nx>=0 && nx<width && ny>=0 && ny<height){
                float weight=kernel[dy*kernel_size+dx];
                if(is_RGB){
                    int nidx=(ny*width+nx)*3;

                    sum_r+=input[nidx]*weight;
                    sum_g+=input[nidx+1]*weight;
                    sum_b+=input[nidx+2]*weight;
                }
                else{
                    int nidx=ny*width+nx;

                    sum_gray+=input[nidx]*weight;
                }
                
            }
        }
    if(is_RGB){
        output[RGB_idx]=static_cast<unsigned char>(max(0,min(static_cast<int>(sum_r/kernel_weight),255)));
        output[RGB_idx+1]=static_cast<unsigned char>(max(0,min(255,static_cast<int>(sum_g/kernel_weight))));
        output[RGB_idx+2]=static_cast<unsigned char>(max(0,min(static_cast<int>(sum_b/kernel_weight),255)));
    }
    else{
        output[grayscale_idx]=static_cast<unsigned char>(max(0,min(static_cast<int>(sum_gray/kernel_weight),255)));
    }    

}
__global__ void edge_detection_kernel(unsigned char *input,unsigned char *output,int width,int height,int direction,int method){
    //TODO handle error for even sized kernels 
    int2 idx=find_index();

    if(idx.x>=width || idx.y>=height)
        return;
    __shared__ float kernel_y[9],kernel_x[9];
    //kernel construction
    if(method==SOBEL){
        //kernel values
        // Sobel x -1   0   1
        //         -2   0   2
        //         -1   0   1
        kernel_x[0]=-1,   kernel_x[1]=0,    kernel_x[2]=1;  
        kernel_x[3]=-2,   kernel_x[4]=0,    kernel_x[5]=2;
        kernel_x[6]=-1,   kernel_x[7]=0,    kernel_x[8]=1;

        // Sobel y -1  -2  -1
        //          0   0   0
        //          1   2   1
        kernel_y[0]=-1,   kernel_y[1]=-2,    kernel_y[2]=-1;  
        kernel_y[3]=0,    kernel_y[4]=0,     kernel_y[5]=0;
        kernel_y[6]=1,    kernel_y[7]=2,     kernel_y[8]=1;

    }
    else if(method==PREWITT){
        //kernel values
        // Prewitt x 1   0  -1
        //           1   0  -1
        //           1   0  -1
        kernel_x[0]=1,   kernel_x[1]=0,    kernel_x[2]=-1;  
        kernel_x[3]=1,   kernel_x[4]=0,    kernel_x[5]=-1;
        kernel_x[6]=1,   kernel_x[7]=0,    kernel_x[8]=-1;

        // Prewitt y  1   1   1
        //            0   0   0
        //           -1  -1  -1
        kernel_y[0]=1,   kernel_y[1]=1,    kernel_y[2]=1;  
        kernel_y[3]=0,   kernel_y[4]=0,    kernel_y[5]=0;
        kernel_y[6]=-1,  kernel_y[7]=-1,   kernel_y[8]=-1;

    }
    else if(method==ROBERT){
        //kernel values
        // Robert x  0   0   0
        //           0   1   0
        //           0   0   -1
        kernel_x[0]=0,   kernel_x[1]=0,    kernel_x[2]=0;  
        kernel_x[3]=0,   kernel_x[4]=1,    kernel_x[5]=0;
        kernel_x[6]=0,   kernel_x[7]=0,    kernel_x[8]=-1;
       // Robert x  0   0   0
        //          0   0   1
        //          0  -1   0
        kernel_y[0]=0,   kernel_y[1]=0,    kernel_y[2]=0;  
        kernel_y[3]=0,   kernel_y[4]=0,    kernel_y[5]=1;
        kernel_y[6]=0,   kernel_y[7]=-1,   kernel_y[8]=0;

    }
    __syncthreads();


    if(direction==1)
        image_kernel_convolution(input,output,width,height,idx.x,idx.y,3,1,kernel_x,false);
    else if(direction==2)
        image_kernel_convolution(input,output,width,height,idx.x,idx.y,3,1,kernel_y,false);




}
__global__  void matrix_magnitude_kernel(unsigned char *destination,unsigned char *matrix_1,unsigned char *matrix_2,int width,int height){
    int2 idx=find_index();
    if(idx.y<height && idx.x<width){
        int index=idx.y*width+idx.x;
        destination[index]=min(255,static_cast<int>(sqrtf(matrix_1[index]*matrix_1[index]+matrix_2[index]*matrix_2[index])));
    }
}
__global__ void edge_sharpening_kernel(unsigned char *input,unsigned char *output,int width,int height,bool is_RGB){
    int2 idx=find_index();

    if(idx.x>=width || idx.y>=height)
        return;

    __shared__ float kernel[9];

    //kernel values
    // Laplacian    0   -1   0
    //             -1    5  -1
    //              0   -1   0
    kernel[0]=0,   kernel[1]=-1,    kernel[2]=0;  
    kernel[3]=-1,   kernel[4]=5,    kernel[5]=-1;
    kernel[6]=0,   kernel[7]=-1,    kernel[8]=0;
    
    __syncthreads();
    

    image_kernel_convolution(input,output,width,height,idx.x,idx.y,3,1,kernel,is_RGB);

    

}